#include <vector>
#include <math.h>

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

/*
static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}
*/

__global__ void mykernel(float* result, float * normalized_data, int nx, int ny) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny)
        return;
    if (j > i) {
        result[i + ny * j] = 0;
        return;
    }
    float sum = 0;
    for (int k = 0; k < nx; k++) {
        sum = sum + normalized_data[k + nx * j] * normalized_data[k + nx * i];
    }
    result[i + ny * j] = sum;
}

void correlate(int ny, int nx, const float *data, float *result) {

    std::vector<float> normalized_data = std::vector<float> (nx * ny, 0);
    for (int j = 0; j < ny; j++) {
        int row_indx = nx * j;
        float mean = 0;
        for (int i = 0; i < nx; i++) {
            mean = mean + data[i + row_indx];
        }
        mean = mean/nx;
        for (int i = 0; i < nx; i++) {
            normalized_data[i + row_indx] = data[i + row_indx] - mean;
        }
        float sqr_sum = 0;
        for (int i = 0; i < nx; i++) {
            sqr_sum = sqr_sum + normalized_data[i + row_indx] * normalized_data[i + row_indx];
        }
        for (int i = 0; i < nx; i++) {
            normalized_data[i + row_indx] = normalized_data[i + row_indx]/sqrt(sqr_sum);
        }
    }

    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, nx * ny * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, &normalized_data[0], nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, nx, ny);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));

}